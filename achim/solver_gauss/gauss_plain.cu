

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define Ae( j , i, N ) (j -1 ) * ( N + 1 ) + i -1

#define ME( A, j , i ) A->elements[ Ae( j , i, A->n ) ]
#define MED( A, j , i ) A->device_elements[ (j -1 ) * ( A->n + 1 ) + i -1 ]



typedef float        t_ve   ;
//#typedef unsigned int t_vidx ; // index of vector elements

t_ve *a;

typedef struct {

    unsigned int n;
    t_ve*    elements;
    t_ve*    x;

    t_ve*    device_elements;
    t_ve*    device_x;

} t_matrix;

typedef t_matrix* t_pmatrix;

t_matrix M1;

// -----------------------------------------------------------------------

__global__ void device_eleminate( t_pmatrix matrix  )
{
    unsigned int i, j, k, max, N;
    t_ve t;

    unsigned int tidx = threadIdx.y * blockDim.x + threadIdx.x;

    N = matrix->n;

    if ( tidx == 0 ) {
    for ( i = 1; i <= N ; i++ ) {
       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( MED( matrix, j , i ) ) > abs( MED( matrix, max , i ) ) ) {
              max = j;
           }
       }
       for ( k = i; k <= N; k++ ) {
          t                     = MED( matrix, i , k );
          ME( matrix, i , k )   = MED( matrix, max , k );
          ME( matrix, max , k ) = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             MED( matrix, j , k ) -= MED( matrix, i , k ) * MED( matrix, j , i ) /  MED( matrix, i , i );
          }
       }
    }
    }
}



// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}
// -----------------------------------------------------------------------
void malloc_matrix( unsigned int size_n, t_pmatrix matrix ) {

   matrix->n        = size_n;
   matrix->elements =  (t_ve*) malloc( sizeof(t_ve) * (size_n + 1 ) * size_n ); /* store b in array, too */
   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
   matrix->x = (t_ve*) malloc( sizeof(t_ve)  * size_n ); /* the output vector */

   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
}




// -----------------------------------------------------------------------
void substitute( t_pmatrix matrix ) {
   unsigned int j, k, N;
   t_ve t;

   N = matrix->n;

   for (j = N; j >= 1; j-- ) {
       t = 0.0;
       for ( k = j + 1; k <= N; k++ ) {
           t +=  ME( matrix, j , k ) * matrix->x[ k - 1 ];
       }
       matrix->x[ j - 1 ] = ( ME( matrix, j , N + 1 ) - t ) / ME( matrix, j , j );
   }
}
// -----------------------------------------------------------------------

void eleminate ( t_ve* Ab, unsigned int N ) {
    unsigned int i, j, k, max;
    t_ve t;


    for ( i = 1; i <= N ; i++ ) {
       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ Ae( j , i , N ) ] ) > abs( Ab[ Ae( max , i, N ) ] )  ) {
              max = j;
           }
       }
       for ( k = i; k <= N; k++ ) {
          t                   = Ab[ Ae(   i , k, N ) ];
          Ab[ Ae( i , k ,  N )   ] = Ab[ Ae( max , k, N ) ];
          Ab[ Ae( max , k, N ) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ Ae( j , k , N ) ] -= Ab[ Ae( i , k, N ) ] * Ab[ Ae( j , i, N ) ] /  Ab[ Ae( i , i, N ) ];
          }
       }
    }
}
// -----------------------------------------------------------------------
void dump_matrix( t_pmatrix matrix ) {
    int n;
    int m;
    for ( m = 0; m < matrix->n; m++ ) {
        printf( "\n  %u. ", m + 1 );
        for ( n = 0; n < matrix->n; n++ ) {
            printf( " %f", matrix->elements[ m * ( matrix->n + 1 ) + n ] );
        }
        printf( " \t b %f", matrix->elements[ m * ( matrix->n + 1 ) + n ] );
   }
   for ( m = 0; m < matrix->n; m++ ) {
      printf( "\n  x%u  = %f",m + 1, matrix->x[ m ] );
   }
}
// -----------------------------------------------------------------------
void gen_textinput_01( t_pmatrix matrix ) {

// Example from R.Sedgewick, Page 608

   malloc_matrix( 3, matrix );

   matrix->elements[ 0 ]  = 1;
   matrix->elements[ 1 ]  = 3;
   matrix->elements[ 2 ]  = -4;

   matrix->elements[ 4 ]  = 1;
   matrix->elements[ 5 ]  = 1;
   matrix->elements[ 6 ]  = -2;

   matrix->elements[  8 ]  = -1;
   matrix->elements[  9 ]  = -2;
   matrix->elements[ 10 ]  = 5;

   matrix->elements[  3 ]  = 8;
   matrix->elements[  7 ]  = 2;
   matrix->elements[ 11 ]  = -1;
}
// -----------------------------------------------------------------------

int main()
{
//    malloc_matrix( 3, &M1 );

    gen_textinput_01( &M1 );

    printf( "hello world , size ist set to %u\n", M1.n );


    dump_matrix( &M1 );
    eleminate( M1.elements, M1.n );
    substitute( &M1 );

    push_problem_to_device( &M1 );

    int block_size = 64;
    dim3 dimBlock(block_size);

    dim3 dimGrid ( 1 );

//    device_eleminate<<<dimGrid,dimBlock>>>( &M1 );



    printf( "\n" );
    dump_matrix( &M1 );
}



